#include <cstdio>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <limits.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#define NUM_CITIES 100
#define MIN_DISTANCE 10
#define MAX_DISTANCE 500
#define NUM_ANTS 100
// #define NUM_ITERATIONS 100 * NUM_CITIES
#define NUM_ITERATIONS 1
#define ALPHA 1
#define BETA 1
#define EVAPORATION_RATE 0.3
#define Q 1
#define MAX_NO_IMPROVEMENT 500

#define cudaCheckError(stmt)                                                   \
  {                                                                            \
    hipError_t err = stmt;                                                    \
    if (err != hipSuccess) {                                                  \
      printf("CUDA failure %s:%d: '%s'\n", __FILE__, __LINE__,                 \
             hipGetErrorString(err));                                         \
      exit(1);                                                                 \
    }                                                                          \
  }

void generateDistanceMatrix(int matrix[NUM_CITIES][NUM_CITIES]) {
  for (int i = 0; i < NUM_CITIES; i++) {
    for (int j = 0; j < NUM_CITIES; j++) {
      if (i == j) {
        matrix[i][j] = 0;
      } else if (j > i) {
        matrix[i][j] =
            MIN_DISTANCE + rand() % (MAX_DISTANCE - MIN_DISTANCE + 1);
        matrix[j][i] = matrix[i][j];
      }
    }
  }
}

void printDistanceMatrix(int matrix[NUM_CITIES][NUM_CITIES]) {
  printf("Distance matrix (in km):\n\n");
  printf("    ");
  for (int i = 0; i < NUM_CITIES; i++) {
    printf("C%-4d", i);
  }
  printf("\n");

  for (int i = 0; i < NUM_CITIES; i++) {
    printf("C%-3d", i);
    for (int j = 0; j < NUM_CITIES; j++) {
      printf("%-4d ", matrix[i][j]);
    }
    printf("\n");
  }
}

void printPheromoneMatrix(double matrix[NUM_CITIES][NUM_CITIES]) {
  printf("Pheromone matrix:\n\n");
  printf("    ");
  for (int i = 0; i < NUM_CITIES; i++) {
    printf("C%-8d", i);
  }
  printf("\n");

  for (int i = 0; i < NUM_CITIES; i++) {
    printf("C%-3d", i);
    for (int j = 0; j < NUM_CITIES; j++) {
      printf("%-4f ", matrix[i][j]);
    }
    printf("\n");
  }
}

__global__ void initPheromonesKernel(float *pheromones_d) {
  int i = blockIdx.x;
  int j = threadIdx.x;

  if (j < NUM_CITIES) {
    pheromones_d[i * NUM_CITIES + j] = 1.0;
  }
}

__device__ void decideNext(float *probabilities, hiprandState state,
                           int *output) {
  float decision = hiprand_uniform(&state);

  float sum = 0;
  for (int i = 0; i < NUM_CITIES; i++) {
    sum += probabilities[i];
    if (sum >= decision) {
      *output = i;
      return;
    }
  }

  *output = NUM_CITIES - 1;
}

void printPath(int path[NUM_CITIES]) {
  for (int i = 0; i < NUM_CITIES; i++) {
    printf("%d", path[i]);
    if (i != NUM_CITIES - 1) {
      printf(" -> ");
    }
  }
}

__global__ void evaporatePheromonesKernel(float *pheromones_d) {
  int i = blockIdx.x;
  int j = threadIdx.y;

  if (j < NUM_CITIES) {
    pheromones_d[i * NUM_CITIES + j] *= (1 - EVAPORATION_RATE);
  }
}

__global__ void addPheromonesKernel(float *pheromones_d, int *antPaths_d,
                                    int *antPathLengths_d) {
  int ant = blockIdx.x;
  int move = threadIdx.x;

  __shared__ double pathLength;
  pathLength = (float)antPathLengths_d[ant];

  if (move < NUM_CITIES - 1) {
    int source = antPaths_d[ant * NUM_CITIES + move];
    int dest = antPaths_d[ant * NUM_CITIES + move + 1];
    float updateVal = Q / pathLength;
    atomicAdd(pheromones_d + source * NUM_CITIES + dest, updateVal);
    atomicAdd(pheromones_d + dest * NUM_CITIES + source, updateVal);
  } else if (move == NUM_CITIES - 1) {
    int source = antPaths_d[ant * NUM_CITIES + move];
    int dest = antPaths_d[ant * NUM_CITIES];
    float updateVal = Q / pathLength;
    atomicAdd(pheromones_d + source * NUM_CITIES + dest, updateVal);
    atomicAdd(pheromones_d + dest * NUM_CITIES + source, updateVal);
  }
}

__global__ void initAntStatesKernel(int *antPaths_d, int *visited_d) {
  int ant = blockIdx.x;
  int i = threadIdx.x;

  if (i < NUM_ANTS) {
    antPaths_d[ant * NUM_CITIES + i] = i == 0 ? ant : 0;
    visited_d[ant * NUM_CITIES + i] = i == ant;
  }
}

__device__ void setProbabilities(float *probabilities, int *visited_d,
                                 float *pheromones_d, int previousCity,
                                 int *distances_d, int ant, int thread,
                                 int move) {
  int nextCity = threadIdx.x;

  if (nextCity >= NUM_CITIES)
    return;

  if (!visited_d[ant * NUM_CITIES + nextCity]) {
    probabilities[nextCity] =
        pow(pheromones_d[previousCity * NUM_CITIES + nextCity], ALPHA) *
        pow((1 / (float)distances_d[previousCity * NUM_CITIES + nextCity]),
            BETA);
  } else {
    probabilities[nextCity] = 0.0;
  }

  __syncthreads();

  float sumProbabilities = 0.0;
  for (int i = 0; i < NUM_CITIES; i++) {
    sumProbabilities += probabilities[i];
  }

  __syncthreads();

  if (ant == 0 && (thread == 96 || thread == 35 || thread == 20) && move == 1) {
    printf("SUM: %f\n", sumProbabilities);
  }

  probabilities[nextCity] /= sumProbabilities;
}

__global__ void antKernel(int *antPaths_d, int *visited_d, float *pheromones_d,
                          int *distances_d, int *antPathLengths_d) {
  int ant = blockIdx.x;
  int pathLenght = 0;

  // if (ant == 0 && threadIdx.x == 0) {
  //   for (int i = 0; i < NUM_CITIES; i++) {
  //     for (int j = 0; j < NUM_CITIES; j++) {
  //       printf("%f ", pheromones_d[i * NUM_CITIES + j]);
  //     }
  //   }
  // }
  for (int move = 1; move < NUM_CITIES; move++) {
    int previousCity = antPaths_d[ant * NUM_CITIES + move - 1];
    __shared__ float probabilities[NUM_CITIES];

    setProbabilities(probabilities, visited_d, pheromones_d, previousCity,
                     distances_d, ant, threadIdx.x, move);

    __syncthreads();

    if (ant == 0 && threadIdx.x == 0 && (move == 1 || move == 2 || move == 3)) {
      float sum = 0.0;
      for (int i = 0; i < NUM_CITIES; i++) {
        sum += probabilities[i];
        printf("%f ", probabilities[i]);
      }
      printf("\n");
      printf("SUM: %f\n", sum);
    }

    if (threadIdx.x == 0) {
      hiprandState d_state;
      hiprand_init(1237, blockIdx.x, 0, &d_state);
      int nextCity;
      decideNext(probabilities, d_state, &nextCity);
      antPaths_d[ant * NUM_CITIES + move] = nextCity;
      visited_d[ant * NUM_CITIES + nextCity] = 1;
      pathLenght += distances_d[previousCity * NUM_CITIES + nextCity];
    }
  }

  if (threadIdx.x == 0) {
    pathLenght +=
        distances_d[antPaths_d[ant * NUM_CITIES + NUM_CITIES - 1] * NUM_CITIES +
                    antPaths_d[ant * NUM_CITIES]];
    antPathLengths_d[ant] = pathLenght;
  }
}

void aco(int distances[NUM_CITIES][NUM_CITIES]) {
  int antPaths[NUM_ANTS][NUM_CITIES];
  int antPathLengths[NUM_ANTS];

  int shortestPathLength = INT_MAX;
  int shortestPath[NUM_CITIES];
  int noImprovement = 0;

  int *antPaths_d, *visited_d, *antPathLengths_d;
  int sizePerAnt = NUM_ANTS * NUM_CITIES * sizeof(int);

  float *pheromones_d;
  int *distances_d;
  int sizePerCityInt = NUM_CITIES * NUM_CITIES * sizeof(int);
  int sizePerCityFloat = NUM_CITIES * NUM_CITIES * sizeof(float);

  // ALLOCATE MEMORY FOR CUDA
  cudaCheckError(hipMalloc((void **)&antPaths_d, sizePerAnt));
  cudaCheckError(hipMalloc((void **)&visited_d, sizePerAnt));
  cudaCheckError(
      hipMalloc((void **)&antPathLengths_d, NUM_ANTS * sizeof(int)));

  cudaCheckError(hipMalloc((void **)&pheromones_d, sizePerCityFloat));
  cudaCheckError(hipMalloc((void **)&distances_d, sizePerCityInt));

  // COPY
  cudaCheckError(hipMemcpy(distances_d, distances, sizePerCityInt,
                            hipMemcpyHostToDevice));

  initPheromonesKernel<<<NUM_CITIES, 128>>>(pheromones_d);

  for (int iterNum = 0; iterNum < NUM_ITERATIONS; iterNum++) {

    initAntStatesKernel<<<NUM_ANTS, 128>>>(antPaths_d, visited_d);

    antKernel<<<NUM_ANTS, 128>>>(antPaths_d, visited_d, pheromones_d,
                                 distances_d, antPathLengths_d);

    cudaCheckError(
        hipMemcpy(antPaths, antPaths_d, sizePerAnt, hipMemcpyDeviceToHost));
    cudaCheckError(hipMemcpy(antPathLengths, antPathLengths_d,
                              NUM_ANTS * sizeof(int), hipMemcpyDeviceToHost));

    int bestAnt = -1;
    for (int ant = 0; ant < NUM_ANTS; ant++) {
      int pathLength = antPathLengths[ant];
      if (pathLength < shortestPathLength) {
        shortestPathLength = pathLength;
        bestAnt = ant;
        noImprovement = 0;
      }
    }
    if (bestAnt != -1) {
      for (int i = 0; i < NUM_CITIES; i++) {
        shortestPath[i] = antPaths[bestAnt][i];
      }
    }

    if (noImprovement > MAX_NO_IMPROVEMENT) {
      printf("Convergence on iter %d\n", iterNum);
      break;
    }

    noImprovement++;

    evaporatePheromonesKernel<<<NUM_CITIES, 128>>>(pheromones_d);
    addPheromonesKernel<<<NUM_ANTS, 128>>>(pheromones_d, antPaths_d,
                                           antPathLengths_d);
  }

  // FREE MEMORY ON DEVICE
  hipFree(antPaths_d);
  hipFree(visited_d);
  hipFree(antPathLengths_d);

  hipFree(pheromones_d);
  hipFree(distances_d);

  printf("Shortest: %d\n", shortestPathLength);
  printPath(shortestPath);
}

int main() {
  int distances[NUM_CITIES][NUM_CITIES];
  generateDistanceMatrix(distances);
  clock_t start = clock();
  aco(distances);
  clock_t end = clock();
  printf("\nCuda time: %f\n", (float)(end - start) / CLOCKS_PER_SEC);

  return 0;
}
